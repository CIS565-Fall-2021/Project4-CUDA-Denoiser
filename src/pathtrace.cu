#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define SHOW_T 0
#define SHOW_POS 1
#define SHOW_NOR 2
#define SHOW_GBUFFER_TYPE 2

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__global__ void sendDenoisedToPBO(uchar4* pbo, glm::ivec2 resolution,
    int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__global__ void gbufferToPBO(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);

        if (SHOW_GBUFFER_TYPE == SHOW_T) {
            float timeToIntersect = gBuffer[index].t * 256.0;

            pbo[index].w = 0;
            pbo[index].x = timeToIntersect;
            pbo[index].y = timeToIntersect;
            pbo[index].z = timeToIntersect;
        }
        
        else if (SHOW_GBUFFER_TYPE == SHOW_POS) {
            glm::vec3 position = glm::normalize(abs(gBuffer[index].pos)) * glm::vec3(256.f);
            pbo[index].w = 0;
            pbo[index].x = position.x;
            pbo[index].y = position.y;
            pbo[index].z = position.z;
        }

        else {
            glm::vec3 normal = abs(gBuffer[index].nor) * glm::vec3(256.f);
            pbo[index].w = 1;
            pbo[index].x = normal.x;
            pbo[index].y = normal.y;
            pbo[index].z = normal.z;
        }
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static glm::vec3* dev_denoised = NULL;
static glm::vec3* dev_denoised_tmp = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
static float* dev_gaussian_kernel = NULL;
static float* dev_gaussian_offsets = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));
    hipMalloc(&dev_denoised, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_denoised, 0, pixelcount * sizeof(glm::vec3));
    hipMalloc(&dev_denoised_tmp, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_denoised_tmp, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

    hipMalloc(&dev_gaussian_kernel, 25 * sizeof(float));
    float kernel[25] = {
        0.003765,	0.015019,	0.023792,	0.015019,	0.003765,
        0.015019,	0.059912,	0.094907,	0.059912,	0.015019,
        0.023792,	0.094907,	0.150342,	0.094907,	0.023792,
        0.015019,	0.059912,	0.094907,	0.059912,	0.015019,
        0.003765,	0.015019,	0.023792,	0.015019,	0.003765, 
    };
    hipMemcpy(dev_gaussian_kernel, &kernel[0], 25 * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&dev_gaussian_offsets, 25 * sizeof(float));
    hipMemset(dev_gaussian_offsets, 0, 25 * sizeof(float));

    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    hipFree(dev_gBuffer);
    hipFree(dev_gaussian_kernel);
    hipFree(dev_gaussian_offsets);
    hipFree(dev_denoised);
    hipFree(dev_denoised_tmp);
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadeSimpleMaterials (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment segment = pathSegments[idx];
    if (segment.remainingBounces == 0) {
      return;
    }

    if (intersection.t > 0.0f) { // if the intersection exists...
      segment.remainingBounces--;
      // Set up the RNG
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.remainingBounces);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        segment.color *= (materialColor * material.emittance);
        segment.remainingBounces = 0;
      }
      else {
        segment.color *= materialColor;
        glm::vec3 intersectPos = intersection.t * segment.ray.direction + segment.ray.origin;
        scatterRay(segment, intersectPos, intersection.surfaceNormal, material, rng);
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      segment.color = glm::vec3(0.0f);
      segment.remainingBounces = 0;
    }

    pathSegments[idx] = segment;
  }
}

__global__ void generateGBuffer (
  int num_paths,
  ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
  GBufferPixel* gBuffer) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
      ShadeableIntersection isect = shadeableIntersections[idx];
      Ray ray = pathSegments[idx].ray;
      gBuffer[idx].t = isect.t;
      gBuffer[idx].pos = ray.origin + glm::vec3(isect.t) * ray.direction;
      gBuffer[idx].nor = isect.surfaceNormal;
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

__device__ float w(GBufferPixel& p, GBufferPixel& q, glm::vec3 colP, glm::vec3 colQ, float colW, float norW, float posW) {
    float w_rt = /*min(exp(-dot(colP, colQ) / colW * colW), 1.0)*/ 1.0;
    float w_n = exp(-glm::length(p.nor - q.nor) / norW * norW);
    float w_x = exp(-glm::length(p.pos - q.pos) / posW * posW);
    return w_rt * w_n * w_x;
}

__global__ void applyATrousFilter(int nPaths, glm::vec3* dst, glm::vec3* prev_iter, PathSegment* paths, 
                                  float resolution, float* kernel, float offset, GBufferPixel* gbuffers,
                                  float colW, float norW, float posW, bool firstIter)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        glm::vec3 cur_color = glm::vec3(0);
        float k = 0;
        PathSegment path = paths[index];

        for (int x = -2; x < 3; ++x) {
            for (int y = -2; y < 3; ++y) {
                int newX = x * offset;
                int newY = y * offset * resolution;
                int l_index = index + newX + newY;

                if (!(l_index < 0 || l_index >= nPaths)) {
                    float h = kernel[(x + 2) + 5 * (y + 2)];
                    float wW = w(gbuffers[index], gbuffers[l_index], path.color, paths[l_index].color, colW, norW, posW);
                    glm::vec3 c = firstIter ? paths[l_index].color : dst[l_index];
                    cur_color += c * h * wW;
                    k += h * wW;
                }

            }
        }
        //if (lastIter) {
        //    dst[index] += cur_color /*/ k*/;
        //}
        //else {
            dst[index] += (cur_color / k) - prev_iter[index];
            prev_iter[index] = cur_color / k;
        //}
    }
}

void denoiseImage(float filterSize, float colW, float norW, float posW) {
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;
    const int blockSize1d = 128;
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;

    //hipMemset(dev_denoised_tmp, 0, pixelcount * sizeof(glm::vec3));
    hipMemcpy(dev_denoised_tmp, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
    hipMemcpy(dev_denoised, dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

    float iterations = floor(log2(filterSize/5.f));
    float offset = 1;
    
    for (int i = 0; i < iterations; ++i) {
        offset = pow(2, i);
        if (i == 0) {
            applyATrousFilter << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_denoised, dev_denoised_tmp, dev_paths, cam.resolution.x,
                dev_gaussian_kernel, offset, dev_gBuffer, colW, norW, posW, true);
        }
        else {
            applyATrousFilter << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_denoised, dev_denoised_tmp, dev_paths, cam.resolution.x,
                dev_gaussian_kernel, offset, dev_gBuffer, colW, norW, posW, false);
        }
    }
    /*offset = pow(2, iterations);
    applyATrousFilter << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_denoised, dev_denoised_tmp, dev_paths, cam.resolution.x,
        dev_gaussian_kernel, offset, dev_gBuffer, colW, norW, posW, true);*/
}


/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter, bool denoise) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Pathtracing Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * NEW: For the first depth, generate geometry buffers (gbuffers)
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally:
    //     * if not denoising, add this iteration's results to the image
    //     * TODO: if denoising, run kernels that take both the raw pathtraced result and the gbuffer, and put the result in the "pbo" from opengl

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

  // Empty gbuffer
  hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));

	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

  bool iterationComplete = false;
	while (!iterationComplete) {

	// tracing
	dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
		depth
		, num_paths
		, dev_paths
		, dev_geoms
		, hst_scene->geoms.size()
		, dev_intersections
		);
	checkCUDAError("trace one bounce");
	hipDeviceSynchronize();

  if (depth == 0) {
    generateGBuffer<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_intersections, dev_paths, dev_gBuffer);
  }

	depth++;

  shadeSimpleMaterials<<<numblocksPathSegmentTracing, blockSize1d>>> (
    iter,
    num_paths,
    dev_intersections,
    dev_paths,
    dev_materials
  );
  iterationComplete = depth == traceDepth;
	}

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather << <numBlocksPixels, blockSize1d >> > (num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // CHECKITOUT: use dev_image as reference if you want to implement saving denoised images.
    // Otherwise, screenshots are also acceptable.
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    gbufferToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_gBuffer);
}

void showImage(uchar4* pbo, int iter, bool denoise) {
const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    if (denoise) {
        sendDenoisedToPBO<<<blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_denoised);
    }
    else {
        sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);
    }
}
