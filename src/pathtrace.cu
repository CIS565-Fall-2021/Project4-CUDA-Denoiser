#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include <thrust/device_vector.h>

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)

#define SORT_MATERIALS false
#define CACHE_FIRST_BOUNCE false
#define DOF false
#define FOCAL_LEN 4.45f
#define ANTIALIASING false

#define SHOW_T 0
#define SHOW_POS 1
#define SHOW_NOR 2
#define SHOW_GBUFFER_TYPE 1

void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
    int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__global__ void gbufferToPBO(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);

        if (SHOW_GBUFFER_TYPE == SHOW_T) {
            float timeToIntersect = gBuffer[index].t * 256.0;

            pbo[index].w = 0;
            pbo[index].x = timeToIntersect;
            pbo[index].y = timeToIntersect;
            pbo[index].z = timeToIntersect;
        }

        else if (SHOW_GBUFFER_TYPE == SHOW_POS) {
            glm::vec3 position = glm::normalize(abs(gBuffer[index].pos)) * glm::vec3(255.f);
            pbo[index].w = 0;
            pbo[index].x = position.x;
            pbo[index].y = position.y;
            pbo[index].z = position.z;
        }

        else {
            glm::vec3 normal = gBuffer[index].nor;
            normal = abs(glm::vec3(normal.x * 255.f, normal.y * 255.f, normal.z * 255.f));
            pbo[index].w = 1;
            pbo[index].x = normal.x;
            pbo[index].y = normal.y;
            pbo[index].z = normal.z;
        }
    }
}


__global__ void sendDenoisedToPBO(uchar4* pbo, glm::ivec2 resolution,
    int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static thrust::device_ptr<PathSegment*> dev_thrust_alive_paths = NULL;
static PathSegment** dev_alive_paths = NULL;
static PathSegment* dev_first_paths = NULL;
static Triangle* dev_triangles = NULL;
static GBufferPixel* dev_gBuffer = NULL;
static float* dev_gaussian_kernel = NULL;
static float* dev_gaussian_offsets = NULL;
static glm::vec3* dev_denoised = NULL;
static glm::vec3* dev_denoised_tmp = NULL;

// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene* scene) {
    hst_scene = scene;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
    hipMalloc(&dev_first_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_alive_paths, pixelcount * sizeof(PathSegment*));
    dev_thrust_alive_paths = thrust::device_ptr<PathSegment*>(dev_alive_paths);

    for (int i = 0; i < scene->geoms.size(); i++) {
        if (scene->geoms[i].type == MESH) {
            hipMalloc(&dev_triangles, scene->geoms[i].numTriangles * sizeof(Triangle));
            hipMemcpy(dev_triangles, scene->geoms[i].triangles, scene->geoms[i].numTriangles * sizeof(Triangle), hipMemcpyHostToDevice);
        }
    }
    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

    hipMalloc(&dev_gaussian_kernel, 25 * sizeof(float));
    float kernel[25] = {
        0.003765,	0.015019,	0.023792,	0.015019,	0.003765,
        0.015019,	0.059912,	0.094907,	0.059912,	0.015019,
        0.023792,	0.094907,	0.150342,	0.094907,	0.023792,
        0.015019,	0.059912,	0.094907,	0.059912,	0.015019,
        0.003765,	0.015019,	0.023792,	0.015019,	0.003765,
    };
    hipMemcpy(dev_gaussian_kernel, &kernel[0], 25 * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&dev_gaussian_offsets, 25 * sizeof(float));
    hipMemset(dev_gaussian_offsets, 0, 25 * sizeof(float));

    hipMalloc(&dev_denoised, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_denoised, 0, pixelcount * sizeof(glm::vec3));
    hipMalloc(&dev_denoised_tmp, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_denoised_tmp, 0, pixelcount * sizeof(glm::vec3));

    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    hipFree(dev_gaussian_kernel);
    hipFree(dev_gaussian_offsets);
    hipFree(dev_denoised);
    hipFree(dev_denoised_tmp);
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

__global__ void generateGBuffer(
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    GBufferPixel* gBuffer) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_paths)
    {
        ShadeableIntersection isect = shadeableIntersections[idx];
        Ray ray = pathSegments[idx].ray;
        gBuffer[idx].t = isect.t;
        gBuffer[idx].pos = ray.origin + glm::vec3(isect.t) * ray.direction;
        gBuffer[idx].nor = isect.surfaceNormal;
    }
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments, PathSegment** aliveSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];
        aliveSegments[index] = &segment;

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, x, y);
        thrust::uniform_real_distribution<float> u01(0, 1);

        // calculate the ray origin
        if (DOF) {
            float aperture = 0.1;
            float sampleX = u01(rng);
            float sampleY = u01(rng);

            // warp pt to disk
            float r = sqrt(sampleX);
            float theta = 2 * 3.14159 * sampleY;
            glm::vec2 res = glm::vec2(cos(theta), sin(theta)) * r;

            segment.ray.origin = cam.position + glm::vec3(res.x, res.y, 0) * aperture;
        }
        else {
            segment.ray.origin = cam.position;
        }

        if (ANTIALIASING) {
            float rand1 = u01(rng);
            float rand2 = u01(rng);

            x = x + rand1 * 2.0;
            y = y + rand2 * 2.0;
        }

        // calculate the ray direction
        if (DOF) {
            float focalLen = FOCAL_LEN;
            float angle = glm::radians(cam.fov.y);
            float aspect = ((float)cam.resolution.x / (float)cam.resolution.y);
            float ndc_x = 1.f - ((float)x / cam.resolution.x) * 2.f;
            float ndc_y = 1.f - ((float)y / cam.resolution.x) * 2.f;

            glm::vec3 ref = cam.position + cam.view * focalLen;
            glm::vec3 H = tan(angle) * focalLen * cam.right * aspect;
            glm::vec3 V = tan(angle) * focalLen * cam.up;
            glm::vec3 target_pt = ref + V * ndc_y + H * ndc_x;
            segment.ray.direction = normalize(target_pt - segment.ray.origin);
        }
        else {
            segment.ray.direction = glm::normalize(cam.view
                - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
                - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
            );

        }

        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
        segment.terminated = false;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth
    , int num_paths
    , PathSegment** pathSegments
    , Geom* geoms
    , int geoms_size
    , ShadeableIntersection* intersections
    , Triangle* triangles
)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = *pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == MESH) {
                t = meshIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside, triangles);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
            //pathSegment.remainingBounces = 0;
        }
        else
        {
            //The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}


// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
    int iter
    , int num_paths
    , ShadeableIntersection* shadeableIntersections
    , PathSegment** pathSegments
    , Material* materials
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) { // if the intersection exists...
          // Set up the RNG
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx]->remainingBounces);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx]->color *= materialColor * material.emittance;
                pathSegments[idx]->terminated = true;
            }
            else {
                // multiply by the albedo color
                pathSegments[idx]->color *= materialColor;

                // find and set next ray direction
                glm::vec3 intersectPt = getPointOnRay(pathSegments[idx]->ray, intersection.t);
                scatterRay(*pathSegments[idx], intersectPt, intersection.surfaceNormal, material, rng);
                pathSegments[idx]->remainingBounces -= 1;
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            pathSegments[idx]->color = glm::vec3(0.0f);
            pathSegments[idx]->terminated = true;
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

__device__ float w(GBufferPixel& p, GBufferPixel& q, glm::vec3 colP, glm::vec3 colQ, float colW, float norW, float posW) {
    float w_rt = min(exp(-dot(colP - colQ, colP - colQ) / colW), 1.0);
    float w_n = exp(-dot(p.nor - q.nor, p.nor - q.nor) / norW);
    float w_x = exp(-dot(p.pos - q.pos, p.pos - q.pos) / posW);
    return w_rt * w_n * w_x;
}

__global__ void applyATrousFilter(int nPaths, glm::vec3* dst, glm::vec3* prev_iter, glm::vec3* beauty,
    float resolution, float* kernel, float offset, GBufferPixel* gbuffers,
    float colW, float norW, float posW, int numIter, bool firstIter)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        glm::vec3 cur_color = glm::vec3(0);
        float k = 0;

        for (int x = -2; x < 3; ++x) {
            for (int y = -2; y < 3; ++y) {
                int newX = x * offset;
                int newY = y * offset * resolution;
                int l_index = index + newX + newY;

                if (!(l_index < 0 || l_index >= nPaths)) {
                    float h = kernel[(x + 2) + 5 * (y + 2)];
                    glm::vec3 cq = firstIter ? beauty[l_index] / (float)numIter : dst[l_index];
                    glm::vec3 cp = firstIter ? beauty[index] / (float)numIter : dst[index];
                    float wW = w(gbuffers[index], gbuffers[l_index], cp, cq, colW, norW, posW);
                    cur_color += cq * h * wW;
                    k += h * wW;
                }

            }
        }
        dst[index] += (cur_color / k) - prev_iter[index];
        prev_iter[index] = cur_color / k;
    }
}

void denoiseImage(float filterSize, float colW, float norW, float posW, int numIter) {

    std::clock_t start2;
    double duration2;
    start2 = std::clock();
    //std::cout << "starting clock at: " << start2 << std::endl;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;
    const int blockSize1d = 128;
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;

    hipMemset(dev_denoised_tmp, 0, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_denoised, 0, pixelcount * sizeof(glm::vec3));

    float iterations = filterSize < 5 ? 0 : floor(log2(filterSize / 5.f));
    float offset = 1;

    for (int i = 0; i < iterations; ++i) {
        offset = pow(2, i);
        if (i == 0) {
            applyATrousFilter << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_denoised, dev_denoised_tmp, dev_image, cam.resolution.x,
                dev_gaussian_kernel, offset, dev_gBuffer, colW, norW, posW, numIter, true);
        }
        else {
            applyATrousFilter << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_denoised, dev_denoised_tmp, dev_image, cam.resolution.x,
                dev_gaussian_kernel, offset, dev_gBuffer, colW, norW, posW, numIter, false);
        }
    }

    // duration2 = (std::clock() - start2) / (double)CLOCKS_PER_SEC;

     //cout.precision(17);
     //std::cout << "ending clock at: " << std::clock() << std::endl;
     //std::cout << "denoise time: " << fixed << duration2 << '\n';
}

// terminates ray if its terminated flag is raised
struct terminateRay {
    __host__ __device__ bool operator()(const PathSegment* ps) {
        return !ps->terminated;
    }
};

// compares materials for sorting
struct compMaterialID : public binary_function<ShadeableIntersection, ShadeableIntersection, bool> {
    __host__ __device__ bool operator()(const ShadeableIntersection& isect1, const ShadeableIntersection& isect2) {
        return isect1.materialId < isect2.materialId;
    }
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter, bool denoise) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;
    bool isFirstIter = iter == 1;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths, dev_alive_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int init_num_paths = dev_path_end - dev_paths;
    int num_paths = init_num_paths;

    bool iterationComplete = false;
    thrust::device_ptr<PathSegment*> endPtr(dev_alive_paths + pixelcount);

    // if not the first iteration, assume the paths have been cached, harvest
    if (CACHE_FIRST_BOUNCE && !ANTIALIASING && !DOF && !isFirstIter) {
        hipMemcpy(dev_paths, dev_first_paths, pixelcount * sizeof(PathSegment), hipMemcpyDeviceToDevice);
        depth++; // start on second bounce now
    }

    while (!iterationComplete) {

        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
            depth
            , num_paths
            , dev_alive_paths
            , dev_geoms
            , hst_scene->geoms.size()
            , dev_intersections
            , dev_triangles
            );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();

        if (depth == 0) {
            generateGBuffer << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_intersections, dev_paths, dev_gBuffer);
        }

        depth++;

        // sort rays by material
        if (SORT_MATERIALS) {
            thrust::device_ptr<PathSegment*> sorted_paths(dev_alive_paths);
            thrust::device_ptr<ShadeableIntersection> sorted_isects(dev_intersections);
            thrust::sort_by_key(sorted_isects, sorted_isects + num_paths, sorted_paths, compMaterialID());
        }

        // shade paths
        shadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
            iter,
            num_paths,
            dev_intersections,
            dev_alive_paths,
            dev_materials
            );

        // if first iteration, cache first bounce
        if (CACHE_FIRST_BOUNCE && !ANTIALIASING && !DOF && isFirstIter && depth == 1) {
            hipMemcpy(dev_first_paths, dev_paths, pixelcount * sizeof(PathSegment), hipMemcpyDeviceToDevice);
        }

        // perform stream compaction
        thrust::device_ptr<PathSegment*> newPathsEnd = thrust::partition(dev_thrust_alive_paths, endPtr, terminateRay());
        endPtr = newPathsEnd;
        num_paths = endPtr - dev_thrust_alive_paths;

        // if reached max depth or if no paths remain, terminate iteration
        if (depth == traceDepth || num_paths == 0) {
            iterationComplete = true;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather << <numBlocksPixels, blockSize1d >> > (init_num_paths, dev_image, dev_paths);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo) {
    const Camera& cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    gbufferToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, dev_gBuffer);
}

void showImage(uchar4* pbo, int iter, bool denoise) {
    const Camera& cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    if (denoise) {
        sendDenoisedToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_denoised);
    }
    else {
        sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);
    }
}