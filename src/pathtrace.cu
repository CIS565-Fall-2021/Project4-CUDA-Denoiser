#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "main.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

#define V_N 1
#define V_P 0

__global__ void gbufferToPBO(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {

    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        float timeToIntersect = gBuffer[index].t * 256.0;
        glm::vec3 n = gBuffer[index].normal;
        glm::vec3 p = gBuffer[index].pos;
        glm::vec3 color(0.f);

#if V_N 
        color = glm::clamp(glm::abs(n * 255.0f), 0.0f, 255.0f);
#elif V_P
        color = glm::clamp(glm::abs(p * 20.0f), 0.0f, 255.0f);
#else
        color = glm::vec3(timeToIntersect);
#endif
        pbo[index].w = 0;
        pbo[index].x = color[0];
        pbo[index].y = color[1];
        pbo[index].z = color[2];
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

// for denoiser
static glm::vec3* dev_denoisedImage = NULL;

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

    // TODO: initialize any extra device memeory you need

    hipMalloc(&dev_denoisedImage, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_denoisedImage, 0, pixelcount * sizeof(glm::vec3));

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    hipFree(dev_gBuffer);
    // TODO: clean up any extra device memory you created
    hipFree(dev_denoisedImage);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);
        float jitteredX = (float)x + u01(rng);
        float jitteredY = (float)y + u01(rng);
        // TODO: implement antialiasing by jittering the ray
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * (jitteredX - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * (jitteredY - (float)cam.resolution.y * 0.5f)
        );

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

__global__ void computeIntersections(
    int depth
    , int num_paths
    , PathSegment* pathSegments
    , Geom* geoms
    , int geoms_size
    , ShadeableIntersection* intersections
)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;
        bool tmp_outside = true;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
                outside = tmp_outside;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            //The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
            intersections[path_index].intersectionPoint = intersect_point;
            intersections[path_index].outside = outside;
        }
    }
}

__global__ void generateGBuffer (int                    num_paths,
                                 ShadeableIntersection* shadeableIntersections,
                                 PathSegment*           pathSegments,
                                 GBufferPixel*          gBuffer)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection& inter = shadeableIntersections[idx];
        PathSegment& pathSeg = pathSegments[idx];
        Ray& r = pathSeg.ray;
        gBuffer[idx].t = inter.t;
        gBuffer[idx].normal = inter.surfaceNormal;
        gBuffer[idx].pos = r.origin + inter.t * r.direction;
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

// passing cam by const& freezes the app???
__global__ void performOneStepATrousFilter(Camera cam, 
    float colorWeight, 
    float normalWeight, 
    float positionWeight, 
    int currStepWidth, 
    GBufferPixel* gBuffer, 
    glm::vec3* image, 
    glm::vec3* denoisedImage)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);

        float h[25] = {
            1.f / 273.f, 4.f / 273.f,  7.f / 273.f,  4.f / 273.f,  1.f / 273.f,
            4.f / 273.f, 16.f / 273.f, 26.f / 273.f, 16.f / 273.f, 4.f / 273.f,
            7.f / 273.f, 26.f / 273.f, 41.f / 273.f, 26.f / 273.f, 7.f / 273.f,
            4.f / 273.f, 16.f / 273.f, 26.f / 273.f, 16.f / 273.f, 4.f / 273.f,
            1.f / 273.f, 4.f / 273.f,  7.f / 273.f,  4.f / 273.f,  1.f / 273.f,
        };

        float cum_w = 0.f;
        glm::vec3 sum{ 0.f };

        glm::vec3 cval = image[index];
        glm::vec3 nval = gBuffer[index].normal;
        glm::vec3 pval = gBuffer[index].pos;

        for (int i = -2; i <= 2; i++)
        {
            for (int j = -2; j <= 2; j++)
            {
                int u = x + currStepWidth * i;
                int v = y + currStepWidth * j;
                if (u < cam.resolution.x && v < cam.resolution.y && u >= 0 && v >= 0)
                {
                    int currIndex = u + (v * cam.resolution.x);

                    // color
                    glm::vec3 ctmp = image[currIndex];
                    glm::vec3 t = cval - ctmp;
                    float dist2 = glm::dot(t, t);
                    float c_w = glm::min(glm::exp(-(dist2) / colorWeight), 1.f);
                    // normal
                    glm::vec3 ntmp = gBuffer[currIndex].normal;
                    t = nval - ntmp;
                    dist2 = glm::max(glm::dot(t, t) / (currStepWidth * currStepWidth), 0.f);
                    float n_w = glm::min(glm::exp(-(dist2) / normalWeight), 1.f);
                    // position
                    glm::vec3 ptmp = gBuffer[currIndex].pos;
                    t = pval - ptmp;
                    dist2 = glm::dot(t, t);
                    float p_w = glm::min(glm::exp(-(dist2) / positionWeight), 1.f);

                    float weight = c_w * n_w * p_w;
                    int hIndex = i + 2 + (j + 2) * 5;
                    sum += ctmp * weight * h[hIndex];
                    cum_w += weight * h[hIndex];
                }
            }
        }
        denoisedImage[index] = sum / cum_w;
    }
}

void denoisePathTracedImage()
{
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    int stepWidth = 1;
    int colorWeight = ui_colorWeight;
    for (int i = 0; i < ui_iterations; i++) {
        performOneStepATrousFilter << <blocksPerGrid2d, blockSize2d >> > (
            cam,
            colorWeight,
            ui_normalWeight,
            ui_positionWeight,
            stepWidth,
            dev_gBuffer,
            dev_image,
            dev_denoisedImage);
        stepWidth *= 2;
        colorWeight *= 0.5f;
        std::swap(dev_denoisedImage, dev_image);
    }

    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
}

__global__ void shadeMaterial(
    int iter
    , int num_paths
    , ShadeableIntersection* shadeableIntersections
    , PathSegment* pathSegments
    , Material* materials
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) { // if the intersection exists...
            // Set up the RNG
            // LOOK: this is how you use thrust's RNG! Please look at
            // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
                pathSegments[idx].remainingBounces = 0;
            }
            else {
                scatterRay(pathSegments[idx],
                    intersection.intersectionPoint,
                    intersection.surfaceNormal,
                    intersection.outside,
                    material,
                    rng,
                    intersection);
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            glm::vec3 unitDirection = glm::normalize(pathSegments[idx].ray.direction);
            float t = 0.5f * (unitDirection[1] + 1.0f);
            pathSegments[idx].color *= (1.0f - t) * glm::vec3(1.0f, 1.0f, 1.0f) + t * glm::vec3(0.5f, 0.7f, 1.0f);
            pathSegments[idx].remainingBounces = 0;
        }
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    bool iterationComplete = false;
    while (!iterationComplete) {
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
            depth
            , num_paths
            , dev_paths
            , dev_geoms
            , hst_scene->geoms.size()
            , dev_intersections
            );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        shadeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
            );

        // Stream compact away all of the terminated paths.
        PathSegment* newEnd = thrust::stable_partition(thrust::device, dev_paths, dev_paths + num_paths, isTerminated());
        num_paths = newEnd - dev_paths;
        iterationComplete = num_paths <= 0;
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_image, dev_paths);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    gbufferToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_gBuffer);
}

void showImage(uchar4* pbo, int iter) {
const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
}
