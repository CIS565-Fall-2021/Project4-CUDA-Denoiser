#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

////Kernel that writes the image to the OpenGL PBO directly.
//__global__ void sendDenoiseToPBO(uchar4* pbo, glm::ivec2 resolution,
//        int iter, glm::vec3* image) {
//    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
//    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
//
//    if (x < resolution.x && y < resolution.y) {
//        int index = x + (y * resolution.x);
//        glm::vec3 pix = image[index];
//
//        glm::ivec3 color;
//        color.x = glm::clamp((int) (pix.x * 255.0), 0, 255);
//        color.y = glm::clamp((int) (pix.y * 255.0), 0, 255);
//        color.z = glm::clamp((int) (pix.z * 255.0), 0, 255);
//
//        // Each thread writes one pixel location in the texture (textel)
//        pbo[index].w = 0;
//        pbo[index].x = color.x;
//        pbo[index].y = color.y;
//        pbo[index].z = color.z;
//    }
//}

__global__ void gbufferToPBO(uchar4* pbo, glm::ivec2 resolution, GBufferPixel* gBuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        //float timeToIntersect = gBuffer[index].t * 256.0;
        pbo[index].w = 0;
        pbo[index].x = gBuffer[index].normal.x * 256.0f;
        pbo[index].y = gBuffer[index].normal.y * 256.0f;
        pbo[index].z = gBuffer[index].normal.z * 256.0f;
    }
}

static Scene * hst_scene = NULL;
static DenoiseSettings * denoiseSettings = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static GBufferPixel* dev_gBuffer = NULL;
static glm::vec3 * dev_dnImage = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

    hipMalloc(&dev_dnImage, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_dnImage, 0, pixelcount * sizeof(glm::vec3));
    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    hipFree(dev_gBuffer);
    hipFree(dev_dnImage); 
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadeSimpleMaterials (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment segment = pathSegments[idx];
    if (segment.remainingBounces == 0) {
      return;
    }

    if (intersection.t > 0.0f) { // if the intersection exists...
      segment.remainingBounces--;
      // Set up the RNG
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.remainingBounces);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        segment.color *= (materialColor * material.emittance);
        segment.remainingBounces = 0;
      }
      else {
        segment.color *= materialColor;
        glm::vec3 intersectPos = intersection.t * segment.ray.direction + segment.ray.origin;
        scatterRay(segment, intersectPos, intersection.surfaceNormal, material, rng);
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      segment.color = glm::vec3(0.0f);
      segment.remainingBounces = 0;
    }

    pathSegments[idx] = segment;
  }
}

__global__ void generateGBuffer (
  int num_paths,
  ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
  GBufferPixel* gBuffer) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    //gBuffer[idx].t = shadeableIntersections[idx].t;
    gBuffer[idx].normal = shadeableIntersections[idx].surfaceNormal;
    gBuffer[idx].position = getPointOnRay(pathSegments[idx].ray, 
                                          shadeableIntersections[idx].t);
  }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

__global__ void denoise(int n, 
						GBufferPixel* gbuff, 
						glm::vec3* image, 
						glm::vec3 * dnImage,
						int step, 
						int imageWidth,
						float normalWeight,
						float posWeight, 
						float colorWeight)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < n)
    {
        glm::vec3 colSum = glm::vec3(0.0f);
        float wSum = 0.0f;
        // hardcode a 5x5 Gaussian filter
        float GaussianFilter[5][5] = { {1,  4, 6,  4,  1},
                                       {4, 16, 24, 16, 4},
                                       {6, 24, 36, 24, 6},
                                       {4, 16, 24, 16, 4},
                                       {1,  4,  6,  4, 1} };

        // a way to convert from 2d pixel space to the 1d pixel array we have
		int uStepIm = 1;
		int vStepIm = imageWidth;

        // the relative offset from the center pixel in the image
        // e.x. -2, -2 is two pixels left and two pixels up in screenspace
        int imStartX = -2;
        int imStartY = -2;


        // store the gbuffer values for the center pixel of our filter
        // i.e. the one we're actually calculating the color for
        glm::vec3 centralNorm = gbuff[index].normal;
        glm::vec3 centralPos = gbuff[index].position;
        glm::vec3 centralCol = image[index];

        // the cell count in 2d, starting in the upper left corner of
        // our 5x5 filter
		for (int y = 0; y < 5; y++) {
			for (int x = 0; x < 5; x++) {
				int imX = (imStartX + x) * uStepIm * step;
				int imY = (imStartY + y) * vStepIm * step;

				// i is the index for 1d representations of our 2d
				// data, i.e. the beauty pass and the gbuffer
				int i = index + imX + imY;
				if (i < 0 || i >= n) {
					// i can be out of bounds along the edges of the image
					continue;
				}

				// get the Gaussian value for this pixel
				float gVal = GaussianFilter[y][x];

				// get the gbuffer values for this pixel
				glm::vec3 nVal = gbuff[i].normal;
				glm::vec3 pVal = gbuff[i].position;
				glm::vec3 cVal = image[i];

				// get the distance of the gbuffer values 
				// from our central pixel
				//glm::vec3 a = centralCol - cVal;
				float nDist = max(glm::length(centralNorm - nVal)/(step*step), 0.0f);
				float pDist = glm::length(centralPos - pVal);// , centralPos - pVal);
				float cDist = glm::length(centralCol - cVal);// , centralCol - cVal);

				// get the weights based on these distances
				// and our input values
				float nw = min(exp(-1.0f * nDist / normalWeight), 1.0f);
				float pw = min(exp(-1.0f * pDist / posWeight), 1.0f);
				float cw = min(exp(-1.0f * cDist / colorWeight), 1.0f);

				// get the overall 
				float w = nw * pw * cw;

				colSum += cVal * w * gVal;
				wSum += w * gVal;
			}
		}

        //bring denoise
        volatile float3 foo = make_float3(colSum.x, colSum.y, colSum.z);
        volatile float3 bar = make_float3(centralCol.x, centralCol.y, centralCol.z);
        dnImage[index] = colSum / wSum;
        //dnImage[index] = colSum / (256.0f * steps);
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Pathtracing Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * NEW: For the first depth, generate geometry buffers (gbuffers)
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally:
    //     * if not denoising, add this iteration's results to the image
    //     * TODO: if denoising, run kernels that take both the raw pathtraced result and the gbuffer, and put the result in the "pbo" from opengl

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

  // Empty gbuffer
  hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));

	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

  bool iterationComplete = false;
	while (!iterationComplete) {

	// tracing
	dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
		depth
		, num_paths
		, dev_paths
		, dev_geoms
		, hst_scene->geoms.size()
		, dev_intersections
		);
	checkCUDAError("trace one bounce");
	hipDeviceSynchronize();

  if (depth == 0) {
    generateGBuffer<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_intersections, dev_paths, dev_gBuffer);
  }

	depth++;

  shadeSimpleMaterials<<<numblocksPathSegmentTracing, blockSize1d>>> (
    iter,
    num_paths,
    dev_intersections,
    dev_paths,
    dev_materials
  );
  iterationComplete = depth == traceDepth;
	}

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////
    if (*hst_scene->state.denoiseSettings->denoise){

        float nWeight = pow(*hst_scene->state.denoiseSettings->normalWeight, 2);
        float pWeight = pow(*hst_scene->state.denoiseSettings->positionWeight, 2);
        float cWeight = pow(*hst_scene->state.denoiseSettings->colorWeight, 2);

        int steps = *hst_scene->state.denoiseSettings->filterSize / 5;
        for (int step = 1; step <= steps; step++) {
			denoise <<<numBlocksPixels, blockSize1d>>>(num_paths, 
													   dev_gBuffer, 
													   dev_image, 
													   dev_dnImage, 
													   step, 
													   cam.resolution.x,
													   nWeight, 
													   pWeight,
													   cWeight);
        }
    }

    // CHECKITOUT: use dev_image as reference if you want to implement saving denoised images.
    // Otherwise, screenshots are also acceptable.
    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something that you can visualize for debugging.
void showGBuffer(uchar4* pbo) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    gbufferToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, dev_gBuffer);
}

void showDenoise(uchar4* pbo, int iter) {
    const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for visualization
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_dnImage);
}

void showImage(uchar4* pbo, int iter) {
const Camera &cam = hst_scene->state.camera;
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);
}
